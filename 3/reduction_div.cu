#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include"timer.h"

#define NUM 10000

__global__ void find_max(int *arr, int *max, int *min){
	int i;
	int idx= blockIdx.x*blockDim.x + threadIdx.x;

	for(i=(*max)/2; ; i/=2){
		if( idx < i ){
			if( arr[idx] < arr[idx+i] )
				arr[idx] = arr[idx+i];
		}
		else{
			if( arr[idx] < arr[idx-i] )
				arr[idx-i] = arr[idx];
		} // Make path divergency

		if( i%2!=0) break;				
	}
	for(int j=0; i<10; i++){
		if(arr[idx] < arr[idx+j])
			arr[idx]=arr[idx+j];
	}

	*min = arr[(*max)/2];
	*max = arr[0];
}

int main(int argc, char *argv[]){
	int *arr;
	int *d_arr, *d_m, *d_min;
	int max=0, i, n, min=0;
	double st, fn;

	srand(time(NULL));

	if( NUM%1024 != 0 )
		n = ((int)(NUM/1024)+1)*1024;


	arr=(int*)calloc(n, sizeof(int));
	for(i =0; i<n; i++)
		arr[i]=rand()%10000;

	hipMalloc((void**)&d_arr, sizeof(int)*n);
	hipMalloc((void**)&d_m, sizeof(int));
	hipMalloc((void**)&d_min, sizeof(int));

	hipMemcpy( d_arr, arr, sizeof(int)*NUM, hipMemcpyHostToDevice);
	hipMemcpy( d_m, &n, sizeof(int), hipMemcpyHostToDevice);

	GET_TIME(st);

	find_max<<<10, 512>>>(d_arr, d_m, d_min);

	hipMemcpy( &max, d_m, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( &min, d_min, sizeof(int), hipMemcpyDeviceToHost);

	GET_TIME(fn);

	printf("%d %d\n", max, min);
	printf("Elapsed Time: %lf\n", fn-st);

	hipFree(d_arr);
	hipFree(d_m);
	free(arr);

	return 0;
}
