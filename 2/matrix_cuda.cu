#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include"timer.h"

#define N 1024

__global__ void matrix_mul(int *a, int *b, int *c){
	// a[][] * b[][] = c[][]
	// c is initiallized with 0

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int sum = 0;

	if( col<N && row<N){
		for(int i=0; i<N; i++){
			sum+=a[row*N+i]*b[i*N+col];
		}
		c[row*N+col]=sum;
	}
}

int main(int argc, char *argv[]){
	int *d_A, *d_B, *d_C;
	int *h_A, *h_B, *h_C;
	int i, j;
	double st, fn;
	dim3 block(4,4);
	dim3 grid(8,8);

	srand(time(NULL));

	h_A=(int*)calloc(N*N, sizeof(int));
	h_B=(int*)calloc(N*N, sizeof(int));
	h_C=(int*)calloc(N*N, sizeof(int));

	for(i=0; i<N; i++){
		for(j=0; j<N; j++){
			h_A[i*N+j]=rand()%10;
			h_B[i*N+j]=rand()%10;
		}
	}// host memory allocation and initialiization
	
	hipMalloc((void**)&d_A, sizeof(int)*N*N);
	hipMalloc((void**)&d_B, sizeof(int)*N*N);
	hipMalloc((void**)&d_C, sizeof(int)*N*N);
 	// device memory allocation
	
	hipMemcpy( d_A, h_A, sizeof(int)*N*N, hipMemcpyHostToDevice);
	hipMemcpy( d_B, h_B, sizeof(int)*N*N, hipMemcpyHostToDevice);
	// copy matrix from host to device

	GET_TIME(st);

	matrix_mul<<<grid,block>>>(d_A, d_B, d_C);

	hipMemcpy( h_C, d_C, sizeof(int)*N*N, hipMemcpyDeviceToHost);
	// copy multiplication result from device to host

	GET_TIME(fn);

	printf("Elapsed Time : %lf\n", fn-st);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
